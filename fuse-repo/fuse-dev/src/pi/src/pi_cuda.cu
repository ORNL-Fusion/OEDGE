// from https://www.olcf.ornl.gov/tutorials/cuda-monte-carlo-pi/
// 03/06/18

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <time.h>
 
//Declare the CUDA kernel
__global__ void kernel(int* count_d, float* randomnums)
{
        int i;
        double x,y,z;
        //Find the overall ID of the thread
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        i = tid;
        int xidx = 0, yidx = 0;
 
        //Do the MonteCarlo!
        xidx = (i+i);
        yidx = (xidx+1);
 
        //Get the random x,y points
        x = randomnums[xidx];
        y = randomnums[yidx];
        z = ((x*x)+(y*y));
 
        if (z<=1)
                count_d[tid] = 1;
        else
                count_d[tid] = 0;
}
 
//Used to check if there are any errors launching the kernel
void CUDAErrorCheck()
{
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
                printf("CUDA error : %s (%d)\n", hipGetErrorString(error), error);
                exit(0);
        }
}
 
//int main(int argc,char* argv[])
//calling C++ from fortran solution from http://theochem.mercer.edu/interlanguage/testdot.cc
extern "C" {
        int picuda_(void);
        }

int picuda_(void)
{
        //NOTE: if threads and/or blocks is changed, niter needs to be changed to reflect
        //that change (niter=threads*blocks)
        int niter = 100000;
        float *randomnums;
        double pi;
        //Allocate the array for the random numbers
        hipMalloc((void**)&randomnums, (2*niter)*sizeof(float));
        //Use CuRand to generate an array of random numbers on the device
        int status;
        hiprandGenerator_t gen;
        status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MRG32K3A);
        status |= hiprandSetPseudoRandomGeneratorSeed(gen, 4294967296ULL^time(NULL));
        status |= hiprandGenerateUniform(gen, randomnums, (2*niter));
        status |= hiprandDestroyGenerator(gen);
        //Check to see if there was any problem launching the CURAND kernels and generating
        //the random numbers on the device
        if (status != HIPRAND_STATUS_SUCCESS)
        {
                printf("CuRand Failure\n");
                exit(EXIT_FAILURE);
        }
 
        //Threads per thread block to be launched
        int threads = 1000;
        //Number of thread blocks launched
        int blocks = 100;
        int* count_d;
        int *count = (int*)malloc(blocks*threads*sizeof(int));
        unsigned int reducedcount = 0;
        //Allocate the array to hold a value (1,0) whether the point in is the circle (1) or not (0)
        hipMalloc((void**)&count_d, (blocks*threads)*sizeof(int));
        CUDAErrorCheck();
        //Launch the kernel
        kernel <<<blocks, threads>>> (count_d, randomnums);
        //Acts as a kind of code Barrier until the kernel is finished. Kernel calls are nonblocking so
        //the code would continue regardless of whether the kernel succeeded or not without the Sync
        hipDeviceSynchronize();
        CUDAErrorCheck();
        //Copy the resulting array back
        hipMemcpy(count, count_d, blocks*threads*sizeof(int), hipMemcpyDeviceToHost);
        int i = 0;
 
        //Reduce array into int
        for(i = 0; i<niter; i++)
                reducedcount += count[i];
 
        //Free the cudaMalloc()'d arrays
        hipFree(randomnums);
        hipFree(count_d);
        free(count);
 
        //Find the ratio
        pi = ((double)reducedcount/niter)*4.0;
        printf("Pi cuda: %f\n", pi);
 
        return 0;
}